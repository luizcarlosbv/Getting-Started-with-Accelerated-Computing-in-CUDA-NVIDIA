#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f
#define BLOCK_SIZE 256

typedef struct { float x, y, z, vx, vy, vz; } Body;

__global__ void bodyForce(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;
        for (int j = 0; j < n; ++j) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;
            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }
        p[i].vx += dt * Fx;
        p[i].vy += dt * Fy;
        p[i].vz += dt * Fz;
    }
}

int main(const int argc, const char** argv) {
    int nBodies = 2<<11;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    const char * initialized_values;
    const char * solution_values;

    if (nBodies == 2<<11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else { // nBodies == 2<<15
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f; // Time step
    const int nIters = 10;  // Simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    hipMallocManaged(&buf, bytes);

    Body *p = (Body*)buf;

    read_values_from_file(initialized_values, buf, bytes);

    double totalTime = 0.0;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 numBlocks((nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        bodyForce<<<numBlocks, threadsPerBlock>>>(p, dt, nBodies);
        hipDeviceSynchronize();

        for (int i = 0 ; i < nBodies; i++) { // integrate position
            p[i].x += p[i].vx*dt;
            p[i].y += p[i].vy*dt;
            p[i].z += p[i].vz*dt;
        }

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    hipFree(buf);
    return 0;
}
